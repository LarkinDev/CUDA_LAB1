#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <ctime>
using namespace  std;

#define CUDA_CHECK_RETURN(value) {                                                                           \
    hipError_t error = value;                                                                               \
                                                                                                             \
    if (error != hipSuccess) {                                                                              \
        fprintf(stderr, "Error %s at line %d at file %s\n", hipGetErrorString(error), __LINE__, __FILE__);  \
        exit(1);                                                                                             \
    }                                                                                                        \
}

#define VECTOR_SIZE (1000000u)
#define BLOCK_SIZE (256)
#define GRID_SIZE ((VECTOR_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE);


__global__ void initBVec(int *data, int length) {
    int i =  blockIdx.x * blockDim.x + threadIdx.x; 
    if (i < length) { 
        data[i] = 1 - length;
    }
}

__global__ void computeCVect(int *vectorA, int *vectorB, int *vectorC, int length) {
    int i =  blockIdx.x * blockDim.x + threadIdx.x;

    if (i < length) {
        vectorC[i] = vectorA[i] - vectorB[i];
    }
}


void task1() {
    int *A_data = (int*) malloc(sizeof(int) * VECTOR_SIZE);
    int *B_data = (int*) malloc(sizeof(int) * VECTOR_SIZE);
    int *C_data = (int*) malloc(sizeof(int) * VECTOR_SIZE);

    for (int i = 0; i < VECTOR_SIZE; i++) {
        A_data[i] = INT_MAX;
        B_data[i] = 1 - VECTOR_SIZE;
        C_data[i] = 0;
    }

    for (int i = 0; i < VECTOR_SIZE; i++) {
        C_data[i] = A_data[i] - B_data[i];
    }

    free(A_data);
    free(B_data);
    free(C_data);
}

void task2() {
    int *A_data = (int*) malloc(sizeof(int) * VECTOR_SIZE);
    int *C_data = (int*) malloc(sizeof(int) * VECTOR_SIZE);

    for (int i = 0; i < VECTOR_SIZE; i++) {
        A_data[i] = INT_MAX;
        C_data[i] = 0;
    }

    int *AD_data, *BD_data, *CD_data;
    CUDA_CHECK_RETURN(hipMalloc(&AD_data, sizeof(int) * VECTOR_SIZE));
    CUDA_CHECK_RETURN(hipMalloc(&BD_data, sizeof(int) * VECTOR_SIZE));
    CUDA_CHECK_RETURN(hipMalloc(&CD_data, sizeof(int) * VECTOR_SIZE));

    CUDA_CHECK_RETURN(hipMemcpy(AD_data, A_data, sizeof(int) * VECTOR_SIZE, hipMemcpyHostToDevice));

    int block_size = BLOCK_SIZE;
    int grid_size = GRID_SIZE
    initBVec<<<grid_size, block_size>>>(BD_data, VECTOR_SIZE);

    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    computeCVect<<<grid_size, block_size>>>(AD_data, BD_data, CD_data, VECTOR_SIZE);

    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    CUDA_CHECK_RETURN(hipMemcpy(C_data, CD_data, sizeof(int) * VECTOR_SIZE, hipMemcpyDeviceToHost));

    hipFree(AD_data);
    hipFree(BD_data);
    hipFree(CD_data);

    free(A_data);
    free(C_data);
}

void task3() {
    int *A_data, *B_data, *C_data;
    CUDA_CHECK_RETURN(hipMallocManaged(&A_data, sizeof(int) * VECTOR_SIZE));
    CUDA_CHECK_RETURN(hipMallocManaged(&B_data, sizeof(int) * VECTOR_SIZE));
    CUDA_CHECK_RETURN(hipMallocManaged(&C_data, sizeof(int) * VECTOR_SIZE));

    for (int i = 0; i < VECTOR_SIZE; i++) {
        A_data[i] = INT_MAX;
        B_data[i] = 1 - VECTOR_SIZE;
        C_data[i] = 0;
    }
    int block_size = BLOCK_SIZE;
    int grid_size = GRID_SIZE
    computeCVect<<<grid_size, block_size>>>(A_data, B_data, C_data, VECTOR_SIZE);

    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    hipFree(A_data);
    hipFree(B_data);
    hipFree(C_data);
}

int main(int, char**) {
    unsigned int start_time =  clock();
    task1();
    unsigned int end_time = clock();
    unsigned int search_time = end_time - start_time;
    cout << search_time;
    //task2();
    //task3();
}